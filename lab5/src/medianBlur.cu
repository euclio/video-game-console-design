#include "hip/hip_runtime.h"
#include "medianBlur.hpp"

const size_t BLOCK_ROWS = 32;
const size_t BLOCK_COLS = 16;
const size_t MEDIAN_LENGTH = 9;

/*
 * Graciously based on
 * http://stackoverflow.com/questions/22315903/cuda-median-filter-implementation-does-not-produce-desired-results
 */
__global__ void blurKernel (
        unsigned char *d_input_img, unsigned char *d_output_img,
        int d_iRows, int d_iCols) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned char window[MEDIAN_LENGTH];

    if (col > d_iCols || row >= d_iRows) {
        return;
    }

    window[0] = (row==0||col==0) ? 0 :                 d_input_img[(row-1)*d_iCols+(col-1)];
    window[1] = (row==0) ? 0 :                         d_input_img[(row-1)*d_iCols+col];
    window[2] = (row==0||col==d_iCols-1) ? 0 :         d_input_img[(row-1)*d_iCols+(col+1)];
    window[3] = (col==0) ? 0 :                         d_input_img[row*d_iCols+(col-1)];
    window[4] =                                        d_input_img[row*d_iCols+col];
    window[5] = (col==d_iCols-1) ? 0 :                 d_input_img[row*d_iCols+(col+1)];
    window[6] = (row==d_iRows-1||col==0) ? 0 :         d_input_img[(row+1)*d_iCols+(col-1)];
    window[7] = (row==d_iRows-1) ? 0 :                 d_input_img[(row+1)*d_iCols+col];
    window[8] = (row==d_iRows-1||col==d_iCols-1) ? 0 : d_input_img[(row+1)*d_iCols+(col+1)];

    // Order elements
    for (unsigned int j = 0; j < 5; ++j) {
        // Find position of minimum element
        unsigned char temp = window[j];
        unsigned int  idx  = j;
        for (unsigned int l = j + 1; l < 9; ++l) {
            if (window[l] < temp) {
                idx = l;
                temp = window[l];
            }
        }

        // Put found minimum element in its place
        window[idx] = window[j];
        window[j] = temp;
    }

    d_output_img[row*d_iCols + col] = window[4];
}

void gpuMedianBlur(const cv::Mat& input, const cv::Mat& output) {
    unsigned char *device_input, *device_output;

    size_t d_ipimgSize = input.step * input.rows;
    size_t d_opimgSize = output.step * output.rows;

    hipMalloc((void**) &device_input, d_ipimgSize);
    hipMalloc((void**) &device_output, d_opimgSize);

    hipMemcpy(device_input, input.data, d_ipimgSize, hipMemcpyHostToDevice);

    dim3 Threads(BLOCK_ROWS, BLOCK_COLS);
    dim3 Blocks(
            (input.cols + Threads.x - 1) / Threads.x,
            (input.rows + Threads.y - 1) / Threads.y);

    blurKernel<<< Blocks, Threads>>>(
            device_input, device_output, input.rows, input.cols);

    hipDeviceSynchronize();

    hipMemcpy(output.data, device_output, d_opimgSize, hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);
}
